#include "hip/hip_runtime.h"

#include "tensor.h"

namespace ts
{
    template<typename T>
    __global__ void add_cu_kernal(T* result, const T* data1, const T* data2, int length){
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < length)
        {
            result[idx] = data1[idx] + data2[idx];
        }
    }

    template<typename T>
    __global__ void sub_cu_kernal(T* result, const T* data1, const T* data2, int length){
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < length)
        {
            result[idx] = data1[idx] - data2[idx];
        }
    }

    template<typename T>
    __global__ void mul_cu_kernal(T* result, const T* data1, const T* data2, int length){
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < length)
        {
            result[idx] = data1[idx] * data2[idx];
        }
    }

    template<typename T>
    __global__ void div_cu_kernal(T* result, const T* data1, const T* data2, int length){
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < length)
        {
            result[idx] = data1[idx] / data2[idx];
        }
    }



    template<typename T>
    __global__ void ein_cu_kernal(T* result, const T* data1, const T* data2, int t1_height, int t1_width, int t2_width) {
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= t1_height*t2_width)
        {
            return;
        }
        size_t row = idx/t2_width;
        size_t col = idx%t2_width;

        T ans = 0;
        for (size_t i = 0; i < t1_width; i++)
        {
            ans+=data1[row*t1_width + i] * data2[i*t2_width+col];
        }
        result[row*t2_width + col] = ans;
    }

    template<typename T>
    Tensor<T> Tensor<T>::cu_add(Tensor<T>& t){
        Tensor<T> result = Tensor<T>(shape);

        if(gpu_t == nullptr){
            gpu();
        }
        if(t.gpu_t == nullptr){
            t.gpu();
        }

        int threadsPerBlock = 1024;
        int numBlocks = (data_length + threadsPerBlock-1) / threadsPerBlock;

        result.gpu();

        add_cu_kernal<<<numBlocks,threadsPerBlock>>>(result.gpu_t ,gpu_t,t.gpu_t,data_length);

        result.cpu();
        result.gpu_free();

        return result;
    }
    template ts::Tensor<double> ts::Tensor<double>::cu_add(ts::Tensor<double>&);

    template<typename T>
    Tensor<T> Tensor<T>::cu_sub(Tensor<T>& t){
        Tensor<T> result = Tensor<T>(shape);

        if(gpu_t == nullptr){
            gpu();
        }
        if(t.gpu_t == nullptr){
            t.gpu();
        }

        int threadsPerBlock = 1024;
        int numBlocks = (data_length + threadsPerBlock-1) / threadsPerBlock;

        result.gpu();

        sub_cu_kernal<<<numBlocks,threadsPerBlock>>>(result.gpu_t ,gpu_t,t.gpu_t,data_length);

        result.cpu();
        result.gpu_free();

        return result;
    }
    template ts::Tensor<double> ts::Tensor<double>::cu_sub(ts::Tensor<double>&);

    template<typename T>
    Tensor<T> Tensor<T>::cu_mul(Tensor<T>& t){
        Tensor<T> result = Tensor<T>(shape);

        if(gpu_t == nullptr){
            gpu();
        }
        if(t.gpu_t == nullptr){
            t.gpu();
        }

        int threadsPerBlock = 1024;
        int numBlocks = (data_length + threadsPerBlock-1) / threadsPerBlock;

        result.gpu();

        mul_cu_kernal<<<numBlocks,threadsPerBlock>>>(result.gpu_t ,gpu_t,t.gpu_t,data_length);

        result.cpu();
        result.gpu_free();

        return result;
    }
    template ts::Tensor<double> ts::Tensor<double>::cu_mul(ts::Tensor<double>&);

    template<typename T>
    Tensor<T> Tensor<T>::cu_div(Tensor<T>& t){
        Tensor<T> result = Tensor<T>(shape);

        if(gpu_t == nullptr){
            gpu();
        }
        if(t.gpu_t == nullptr){
            t.gpu();
        }

        int threadsPerBlock = 1024;
        int numBlocks = (data_length + threadsPerBlock-1) / threadsPerBlock;

        result.gpu();

        div_cu_kernal<<<numBlocks,threadsPerBlock>>>(result.gpu_t ,gpu_t,t.gpu_t,data_length);

        result.cpu();
        result.gpu_free();

        return result;
    }
    template ts::Tensor<double> ts::Tensor<double>::cu_div(ts::Tensor<double>&);

    template<typename T>
    Tensor<T> Tensor<T>::cu_ein(Tensor<T>& t){
        std::vector<int> shap = {shape[0],t.shape[1]};
        Tensor<T> result = Tensor<T>(shap);
        if(gpu_t == nullptr){
            gpu();
        }
        if(t.gpu_t == nullptr){
            t.gpu();
        }

        int threadsPerBlock = 1024;
        int numBlocks = (data_length + threadsPerBlock-1) / threadsPerBlock;

        result.gpu();

        ein_cu_kernal<<<numBlocks,threadsPerBlock>>>(result.gpu_t,gpu_t,t.gpu_t,shape[0],shape[1],t.shape[1]);

        result.cpu();
        result.gpu_free();

        return result;

    }
    template ts::Tensor<double> ts::Tensor<double>::cu_ein(ts::Tensor<double>&);



    template<typename T>
    void Tensor<T>::gpu(){
        if(gpu_t != nullptr){
            hipFree(gpu_t);
            gpu_t = nullptr;
        }
        hipMalloc((void**)&gpu_t, data_length*sizeof(T)); 
        hipMemcpy(gpu_t, data.get(), data_length*sizeof(T), hipMemcpyHostToDevice);
    }
    template void Tensor<double>::gpu();


    template<typename T>
    void Tensor<T>::cpu(){
        hipMemcpy(data.get(), gpu_t, data_length*sizeof(T),hipMemcpyDeviceToHost);
    }
    template void Tensor<double>::cpu();


    template<typename T>
    void Tensor<T>::gpu_free(){
        if(gpu_t != nullptr){
            hipFree(gpu_t);
            gpu_t = nullptr;
        }
    }
    template void Tensor<double>::gpu_free();


    template<typename T>
    T* Tensor<T>::get_gpu_t(){
        return gpu_t;
    }
    template double* Tensor<double>::get_gpu_t();

    
}